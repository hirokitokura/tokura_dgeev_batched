
#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"tokura_blas_define.h"
#include"tokura_blas_const.h"

#include"tokura_blas.h"
#include"tokura_blas_functions.h"
#include<float.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


int tokura_dgeev_batched_SWB_tune(tokuraInternalhandle_t* tokurahandle, int n, double** A, double** wr, double** wi, int batchCount, float* time)
{
	int i;
	int batchCount_per_stream;
	const int WARPSIZE = tokurahandle->WARP_SIZE;

	int shift;

	double *comp_real_h, *comp_imag_h;


	int tmp_thred_per_matrix;

	int hessen_join_num;
	int matrix_num_per_block;
	hessen_join_num = get_hessenbergreduction_SWBthreads_num(n);


	int doubleqr_join_num = get_doubleshiftQR_MWBthreads_num(n);


	int optimal_hrd_thread = 0;

	int matrix_index = 0;
	int stream_id = 0;



	i = 0;
	int thread_count;
	hipEvent_t start, stop;
	float elapsed_time_ms = 0.0f;
	float hrd_local_time = FLT_MAX;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	for (thread_count = 1; thread_count <= n; thread_count++)
	{
		tmp_thred_per_matrix = thread_count;
		matrix_num_per_block = WARPSIZE / tmp_thred_per_matrix;

		if (
			tokurahandle->sharedsize
			<
			(sizeof(double)*n*n*matrix_num_per_block + sizeof(double)*n*matrix_num_per_block + sizeof(double)*tmp_thred_per_matrix*matrix_num_per_block)
			)
		{
			continue;
		}
		hessen_join_num = tmp_thred_per_matrix;
		for (i = 10; i > 0; i--)
		{
			if ((hessen_join_num >> i) == 1)
			{
				break;
			}
		}

		if ((1 << (i)) == hessen_join_num)
		{
			i = i - 1;
		}
		if (i < 0)
		{
			i = 0;
		}
		if (hessen_join_num == 1)
		{
			i = 0;
		}
		hessen_join_num = 1 << i;
		stream_id = 0;
		batchCount_per_stream = batchCount;

		for (int transferedmatrixid = 0; transferedmatrixid < batchCount_per_stream; transferedmatrixid++)
		{
			hipMemcpyAsync(
				&tokurahandle->A_tmp_device[stream_id][(matrix_index + transferedmatrixid)*n*n],
				A[matrix_index + transferedmatrixid],
				sizeof(double)*n*n,
				hipMemcpyHostToDevice,
				0
			);
		}
		hipDeviceSynchronize();
		hipEventRecord(start, 0);
		dim3 grid, block;
		int temp = WARPSIZE / n;
		block = dim3(WARPSIZE, 4);
		tokura_matrixrearrangement_MWtoRW
			<< <batchCount_per_stream, block, 0 >> >
			(
				n,
				tokurahandle->A_tmp_device[stream_id],
				tokurahandle->A_device[stream_id],
				batchCount_per_stream
				);
		hipDeviceSynchronize();

		grid = dim3((batchCount_per_stream + matrix_num_per_block - 1) / matrix_num_per_block);
		block = dim3(tmp_thred_per_matrix*matrix_num_per_block);
		tokura_dgehrd_batched_SWB
			<< <
			grid,
			block,
			sizeof(double)*n*n*matrix_num_per_block 
			+ sizeof(double)*n*matrix_num_per_block 
			+ sizeof(double)*tmp_thred_per_matrix*matrix_num_per_block
			>> >
			(
				n,
				batchCount_per_stream,
				tokurahandle->A_device[stream_id],
				hessen_join_num,
				tmp_thred_per_matrix,
				matrix_num_per_block
				);
		hipDeviceSynchronize();

		dim3 block_RWtoEW = dim3((batchCount_per_stream + WARPSIZE - 1) / WARPSIZE, n);
		dim3 thread_RWtoEW = dim3(WARPSIZE, 1);
		tokura_matrixrearrangement_RWtoEW
			<< <
			block_RWtoEW,
			thread_RWtoEW,
			0
			>> >
			(
				n,
				batchCount_per_stream,
				tokurahandle->A_device[stream_id],
				tokurahandle->A_tmp_device[stream_id]
				);


		hipDeviceSynchronize();

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsed_time_ms, start, stop);
		if (hrd_local_time > elapsed_time_ms)
		{
			hrd_local_time = elapsed_time_ms;
			optimal_hrd_thread = thread_count;
		}



	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	time[n] = hrd_local_time;
	return optimal_hrd_thread;

}