#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"tokura_blas_define.h"
#include"tokura_blas_const.h"

#include"tokura_blas.h"
#include"tokura_blas_functions.h"


//DGEHRD reduces a DOUBLE PRECISION general matrix A 
//to upper Hessenberg form H by an orthogonal similarity transformation
__global__ void tokura_dgehrd_batched_SWB
(
	const int n_o,
	const  int mat_num,
	double*  a_o,
	const int hessen_join_num,
	const int THREADS_PER_MATRIX,
	const int MATRIX_PER_BLOCK
)
{

	const int thin = (threadIdx.x%THREADS_PER_MATRIX);
	const int mat_pad_index = (threadIdx.x / THREADS_PER_MATRIX);
	const int mat_pad = (threadIdx.x / THREADS_PER_MATRIX)*n_o;
	const int mat_pad_g = (blockIdx.x *  MATRIX_PER_BLOCK) *(n_o);
	const int n = (n_o*MATRIX_PER_BLOCK);
	const int REDUCTION_JOIN_THREADS_NUM = hessen_join_num;

	int i, j, k;
	double tmp;
	double tmp_sum;
	double tau;
	double vex_x_norm;


	if (!(((blockIdx.x *MATRIX_PER_BLOCK) + mat_pad_index) < mat_num))
	{
		return;
	}


	volatile extern __shared__ double SHARED_EXTERN[];
	volatile double *a = &SHARED_EXTERN[0];
	volatile double *house_tmp = &SHARED_EXTERN[n_o*n];
	volatile double *house = &house_tmp[mat_pad];
	volatile double *sum_tmp_shared = &SHARED_EXTERN[n_o*n + n_o*MATRIX_PER_BLOCK];


	const int ALIVE_THREADS_NUM = (mat_num - (blockIdx.x *MATRIX_PER_BLOCK)) > MATRIX_PER_BLOCK ? MATRIX_PER_BLOCK*THREADS_PER_MATRIX : (mat_num - (blockIdx.x *MATRIX_PER_BLOCK))*THREADS_PER_MATRIX;
	const int ALIVE_MATRIX_NUM = (mat_num - (blockIdx.x *MATRIX_PER_BLOCK)) > MATRIX_PER_BLOCK ? MATRIX_PER_BLOCK : (mat_num - (blockIdx.x *MATRIX_PER_BLOCK));


	{
		int count;
		int mat_pad_index;
		for (j = 0; j < n_o; j++)
		{
			for (count = threadIdx.x; count < n_o*ALIVE_MATRIX_NUM; count += ALIVE_THREADS_NUM)
			{
				i = count%n_o;
				mat_pad_index = count / n_o;
				a[j*n + i + mat_pad_index*n_o] = a_o[j*n_o*mat_num + i + mat_pad_index*n_o + mat_pad_g];
			}
		}
	}
	__syncwarp();


	for (k = 1; k < n_o - 1; k++)
	{
		tmp_sum = 0.0;

		i = k + 1 + thin;

		while (i < n_o)
		{
			tmp = a[(k - 1)*n + i + mat_pad];
			house[i] = tmp;
			tmp_sum += tmp*tmp;
			i += THREADS_PER_MATRIX;
		}
		__syncwarp();



		sum_tmp_shared[thin + mat_pad_index*THREADS_PER_MATRIX] = tmp_sum;
		__syncwarp();

		if (thin == 0)
		{
			house[k] = a[((k - 1)*n + (k)) + mat_pad];
		}
		__syncwarp();


		tmp_sum = 0.0;
		i = REDUCTION_JOIN_THREADS_NUM;
		while (i != 0)
		{
			if ((thin < i) && (thin + i < (THREADS_PER_MATRIX)))
			{
				sum_tmp_shared[(thin)+mat_pad_index*THREADS_PER_MATRIX] += sum_tmp_shared[(thin + i) + mat_pad_index*THREADS_PER_MATRIX];
			}
			__syncwarp();

			i = i >> 1;
		}
		__syncwarp();


		tmp_sum = sum_tmp_shared[0 + mat_pad_index*THREADS_PER_MATRIX];
		__syncwarp();

		int FLAG = (tmp_sum == 0.0);
		if (!FLAG)
		{
			if (thin == 0)
			{
				tmp = house[k] * house[k] + tmp_sum;
				tmp = sqrt(tmp);
				vex_x_norm = tmp;
				house[k] += (house[k] > 0.0) ? tmp : -tmp;
			}
		}
		__syncwarp();




		if (!FLAG)
		{
			tmp = house[k] * house[k] + tmp_sum;
		}
		__syncwarp();

		if (!FLAG)
		{
			tmp_sum = tmp;
			if (tmp_sum != 0.0)
			{
				tau = 2.0 / tmp;
			}
		}
		__syncwarp();



		if (!FLAG)
		{
			if (thin == 0)
			{
				a[((k - 1)*n + k) + mat_pad] = (house[k] > 0.0) ? -vex_x_norm : vex_x_norm;
				for (i = k + 1; i < n_o; i++)
				{
					a[((k - 1)*n + i) + mat_pad] = 0.0;
				}
			}
		}
		__syncwarp();

		if (!FLAG)
		{
			//matrix multication from left at similarity transformation

			j = k - 1 + 1 + thin;
			while (j < n_o)
			{
				tmp = 0.0;
				for (i = k; i < n_o; i++)
				{
					tmp += house[i] * a[(j*n + i) + mat_pad];
				}
				tmp *= tau;
				for (i = k; i < n_o; i++)
				{
					a[(j*n + i) + mat_pad] -= house[i] * tmp;
				}
				j += THREADS_PER_MATRIX;


			}
		}
		__syncwarp();

		if (!FLAG)
		{
			//matrix multication from right at similarity transformation

			i = 0 + thin;
			while (i < n_o)
			{
				tmp = 0.0;

				for (j = k; j < n_o; j++)
				{
					tmp += house[j] * a[(j*n + i) + mat_pad];
				}
				tmp *= tau;
				for (j = k; j < n_o; j++)
				{
					a[(j*n + i) + mat_pad] -= house[j] * tmp;
				}
				i += THREADS_PER_MATRIX;
			}
		}
	}
	__syncwarp();



	//__syncwarp();
	{
		int count;
		int mat_pad_index;
		for (j = 0; j < n_o; j++)
		{
			for (count = threadIdx.x; count < n_o*ALIVE_MATRIX_NUM; count += ALIVE_THREADS_NUM)
			{
				i = count%n_o;
				mat_pad_index = count / n_o;


				a_o[j*n_o*mat_num + i + mat_pad_index*n_o + mat_pad_g] = a[j*n + i + mat_pad_index*n_o];
			}
		}
	}

}