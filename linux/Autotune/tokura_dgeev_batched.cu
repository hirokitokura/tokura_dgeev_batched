#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include"tokura_blas_define.h"
#include"tokura_blas_const.h"
//#include"tokura_tuned_thread_parameters.h"


int tokura_dgeev_batched_MWB(tokuraInternalhandle_t* tokurahandle, int n, double** A, double** wr, double** wi, int batchCount);
int tokura_dgeev_batched_SWBMWB(tokuraInternalhandle_t* tokurahandle, int n, double** A, double** wr, double** wi, int batchCount);

int tokura_dgeev_batched_MWB_tune(tokuraInternalhandle_t* tokurahandle, int n, double** A, double** wr, double** wi, int batchCount, float* time);
int tokura_dgeev_batched_SWB_tune(tokuraInternalhandle_t* tokurahandle, int n, double** A, double** wr, double** wi, int batchCount,float* time);


void tokuraCreate(tokuraInternalhandle_t** tokurahandle)
{
	*tokurahandle = (tokuraInternalhandle_t*)malloc(sizeof(tokuraInternalhandle_t));
	hipDeviceProp_t dev;
	hipGetDeviceProperties(&dev, 0);
	(*tokurahandle)->WARP_SIZE = dev.warpSize;
	(*tokurahandle)->sharedsize = dev.sharedMemPerBlock;

}
void tokuraDestroy(tokuraInternalhandle_t* tokurahandle)
{
	free(tokurahandle);
}
void tokuraMemorymalloc(tokuraInternalhandle_t* tokurahandle, int n, int batchCount)
{
	int i;
	
	tokurahandle->A_device = (double**)malloc(sizeof(double*)*NUMBER_OF_STREAMS);
	tokurahandle->A_tmp_device = (double**)malloc(sizeof(double*)*NUMBER_OF_STREAMS);
	tokurahandle->wr_device = (double**)malloc(sizeof(double*)*NUMBER_OF_STREAMS);
	tokurahandle->wr_tmp_device = (double**)malloc(sizeof(double*)*NUMBER_OF_STREAMS);
	tokurahandle->wi_device = (double**)malloc(sizeof(double*)*NUMBER_OF_STREAMS);
	tokurahandle->wi_tmp_device =(double**)malloc(sizeof(double*)*NUMBER_OF_STREAMS);
	tokurahandle->stream = (hipStream_t*)malloc(sizeof(hipStream_t)*NUMBER_OF_STREAMS);

	for (i = 0; i < NUMBER_OF_STREAMS; i++)
	{
		hipMalloc((void**)&tokurahandle->A_device[i], sizeof(double)*n*n*NUMBER_OF_COMPUTED_MATRICES_PER_STREAM);
		hipMalloc((void**)&tokurahandle->A_tmp_device[i], sizeof(double)*n*n*NUMBER_OF_COMPUTED_MATRICES_PER_STREAM);
		hipMalloc((void**)&tokurahandle->wr_device[i], sizeof(double)*n*NUMBER_OF_COMPUTED_MATRICES_PER_STREAM);
		hipMalloc((void**)&tokurahandle->wi_device[i], sizeof(double)*n*NUMBER_OF_COMPUTED_MATRICES_PER_STREAM);
		hipMalloc((void**)&tokurahandle->wr_tmp_device[i], sizeof(double)*n*NUMBER_OF_COMPUTED_MATRICES_PER_STREAM);
		hipMalloc((void**)&tokurahandle->wi_tmp_device[i], sizeof(double)*n*NUMBER_OF_COMPUTED_MATRICES_PER_STREAM);

		hipStreamCreate(tokurahandle->stream + i);
	}

}

void tokuraMemoryfree(tokuraInternalhandle_t* tokurahandle, int n, int batchCount)
{
	int i;


	for (i = 0; i < NUMBER_OF_STREAMS; i++)
	{
		hipFree(tokurahandle->A_device[i]);
		hipFree(tokurahandle->A_tmp_device[i]);
		hipFree(tokurahandle->wr_device[i]);
		hipFree(tokurahandle->wi_device[i]);
		hipFree(tokurahandle->wr_tmp_device[i]);
		hipFree(tokurahandle->wi_tmp_device[i]);
		hipStreamDestroy(tokurahandle->stream[i]);
	}

	free(tokurahandle->A_device);
	free(tokurahandle->A_tmp_device);
	free(tokurahandle->wr_device);
	free(tokurahandle->wr_tmp_device);
	free(tokurahandle->wi_device);
	free(tokurahandle->wi_tmp_device);
	free(tokurahandle->stream);

}


int tokura_dgeev_batched_MWBtune(int n, double** A, double** wr, double** wi, int batchCount, float* time)
{
	if (!(n > 0 && n <= 32))
	{
		//���͍s��T�C�Y�`�F�b�N
		return -1;
	}
	if (A == NULL)
	{
		//���͍s��̃������m�ۃ`�F�b�N
		return -2;
	}
	if (wr == NULL)
	{
		//�o�͌ŗL�l�����̃������m�ۃ`�F�b�N
		return -3;
	}
	if (wi == NULL)
	{
		//�o�͌ŗL�l�����̃������m�ۃ`�F�b�N
		return -4;
	}


	tokuraInternalhandle_t* tokurahandle;
	int threadnum;
	tokuraCreate(&tokurahandle);
	tokuraMemorymalloc(tokurahandle, n, batchCount);
	threadnum =tokura_dgeev_batched_MWB_tune(tokurahandle, n, A, wr, wi, batchCount,time);
	tokuraMemoryfree(tokurahandle, n, batchCount);


	tokuraDestroy(tokurahandle);
	return threadnum;//success

}


int tokura_dgeev_batched_SWBtune(int n, double** A, double** wr, double** wi, int batchCount, float* time)
{
	if (!(n > 0 && n <= 32))
	{
		//���͍s��T�C�Y�`�F�b�N
		return -1;
	}
	if (A == NULL)
	{
		//���͍s��̃������m�ۃ`�F�b�N
		return -2;
	}
	if (wr == NULL)
	{
		//�o�͌ŗL�l�����̃������m�ۃ`�F�b�N
		return -3;
	}
	if (wi == NULL)
	{
		//�o�͌ŗL�l�����̃������m�ۃ`�F�b�N
		return -4;
	}


	tokuraInternalhandle_t* tokurahandle;
	int threadnum;
	tokuraCreate(&tokurahandle);
	tokuraMemorymalloc(tokurahandle, n, batchCount);
	threadnum = tokura_dgeev_batched_SWB_tune(tokurahandle, n, A, wr, wi, batchCount, time);
	tokuraMemoryfree(tokurahandle, n, batchCount);


	tokuraDestroy(tokurahandle);
	return threadnum;//success

}