#ifndef __TOKURADGEEV_BATCHED__
#define __TOKURADGEEV_BATCHED__

#include<stdio.h>
#include<stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "tokura_dgeev_batched_define.h"
#include "tokura_dgeev_batched_const.h"
#include "tokura_dgeev_batched_tuned_thread_parameters.h"

#include "tokura_dgeev_batched_MWB.cuh"
#include "tokura_dgeev_batched_SWBMWB.cuh"
#include "tokura_dgeev_batched_handler.cuh"


#ifdef WIN64
__declspec(dllexport)int tokura_dgeev_batched(int n, double** A, double** wr, double** wi, int batchCount, char* flags);
__declspec(dllexport)void tokura_malloc(double **A, int size, int  batchCount);
__declspec(dllexport)void tokura_flags_malloc(char** flags, int batchCount);
__declspec(dllexport)void tokura_free(double** A);
__declspec(dllexport)void tokura_flags_free(char* flags);
__declspec(dllexport)void tokura_cudaHostRegister(double** mat, int size);
__declspec(dllexport)void tokura_cudaHostUnregister(double* mat);
#endif
#ifdef __unix__
int tokura_dgeev_batched(int n, double** A, double** wr, double** wi, int batchCount,char* flags);
void tokura_malloc(double **A, int size, int  batchCount);
void tokura_flags_malloc(char** flags, int batchCount);
void tokura_free(double** A);
void tokura_flags_free(char* flags);
void tokura_cudaHostRegister(double** mat, int size);
void tokura_cudaHostUnregister(double* mat);

#endif
//For MATLAB
void tokura_cudaHostRegister(double** mat, int size)
{
	hipHostRegister(*mat, sizeof(double)*size, hipHostMallocDefault);
}
//For MATLAB
void tokura_cudaHostUnregister(double* mat)
{
	hipHostUnregister(mat);
}
//Host memory allocation for matrices or eigenvalues
void tokura_malloc(double **A, int size, int  batchCount)
{
	int i;
	double** A_pointer = A;
	double *tmp_pointer;
	hipHostMalloc(&tmp_pointer, size*batchCount * sizeof(double));
	for (i = 0; i < batchCount; i++)
	{
		//	hipHostMalloc(&A[i], size* sizeof(double));

		A[i] = &tmp_pointer[i*size];
	}
}
//Host memory allocation for flags
void tokura_flags_malloc(char** flags, int batchCount)
{
	hipHostMalloc(flags, batchCount * sizeof(char));

}
//Host memory free for matrices or eigenvalues
void tokura_free(double** A)
{

	hipHostFree(A[0]);

}

//Host memory free for flags
void tokura_flags_free(char* flags)
{
	hipHostFree(flags);
}

//Helper function
//This function switch thread assginents 
//Switching is depended on matrix size.
int tokura_dgeev_batched(int n, double** A, double** wr, double** wi,int batchCount,char* flags)
{
	if (!(n > 0 && n<= MAX_MATRIX_SIZE))
	{
		return -1;
	}
	if (A == NULL)
	{
		return -2;
	}
	if (wr == NULL)
	{
		return -3;
	}
	if (wi == NULL)
	{
		return -4;
	}
	if (!(batchCount > 0))
	{
		return -5;
	}
	if (flags == NULL)
	{
		return -6;
	}


	tokuraInternalhandle_t* tokurahandle;
	tokuraCreate(&tokurahandle);
	tokuraMemorymalloc(tokurahandle, n, batchCount);

	int i;
	

	if (n < TOKURA_SWITCH_ALGORITHM_MATRIXSIZE)
	{
		tokura_dgeev_batched_MWB(tokurahandle, n, A, wr, wi, batchCount, flags);
	}
	else
	{
		tokura_dgeev_batched_SWBMWB(tokurahandle, n, A, wr, wi, batchCount, flags);
	}

	tokuraMemoryfree(tokurahandle, n, batchCount);

	tokuraDestroy(tokurahandle);


	return 0;
}
#endif