
#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include"tokura_blas_define.h"
//#include"const.h"

#include<omp.h>

#include"MT.h"

#include"tokura_blas.h"
#define MAX_MATRIX_SIZE 32

#define USE_MATRIX_NUM_FOR_TUNE (32768>>2)

void set_mat(double **A, int n, int batchCount)
{
	int i = 0;
	int j, k;
	double tmp;
	FILE* fp;


	init_genrand(100);

	for (k = 0; k < batchCount; k++)
	{
		for (i = 0; i <n; i++)
		{
			for (j = 0; j <n; j++)
			{
				A[k][j*n + i] = genrand_res53();

			}
		}
	}

	return;
}
void parameter_write(float time[2][MAX_MATRIX_SIZE + 1], int threadnum[2][MAX_MATRIX_SIZE * 2])
{
	int switch_size = MAX_MATRIX_SIZE + 1;
	int n;
	for (n = 1; n <= MAX_MATRIX_SIZE; n++)
	{
		if (time[0][n] > time[1][n])
		{
			switch_size = n + 1;
			break;
		}
	}
	printf("SWITCH %d\n", switch_size);

	char define_name[1024];
	int i;
	FILE* fp;
	fp = fopen("../tokura_dgeev_batched/tokura_dgeev_batched_tuned_thread_parameters.h", "w");
	if (fp == NULL)
	{
		printf("FILE OPEN ERROR\n");
		exit(-1);
	}

	fprintf(fp, "#ifndef __TOKURABLAS_TUNED_PARAMETERS__\n");
	fprintf(fp, "#define __TOKURABLAS_TUNED_PARAMETERS__\n");
	fprintf(fp, "#define NUMBER_OF_COMPUTED_MATRICES_PER_STREAM %d\n", USE_MATRIX_NUM_FOR_TUNE);
	fprintf(fp, "#define TOKURA_SWITCH_ALGORITHM_MATRIXSIZE %d\n", switch_size);


	for (i = 1; i <= MAX_MATRIX_SIZE; i++)
	{
		sprintf(define_name, "#define TOKURA_MWB_HRD_%d %d\n", i, threadnum[0][i]);
		fprintf(fp, "%s", define_name);
	}
	for (i = 1; i <= MAX_MATRIX_SIZE; i++)
	{
		sprintf(define_name, "#define TOKURA_MWB_DOUBLESHIFTQR_%d %d\n", i, threadnum[0][i + MAX_MATRIX_SIZE]);
		fprintf(fp, "%s", define_name);
	}
	for (i = 1; i <= MAX_MATRIX_SIZE; i++)
	{
		sprintf(define_name, "#define TOKURA_SWB_HRD_%d %d\n", i, threadnum[1][i]);
		fprintf(fp, "%s", define_name);
	}
	fprintf(fp, "#endif\n");

	fclose(fp);

}


int main(void)
{
	int i;
	int n;//n: �s��T�C�Y
	int batchCount = USE_MATRIX_NUM_FOR_TUNE;
		;//batchCount: �s��
	int info;

	float time[2][MAX_MATRIX_SIZE+1];

	int threadnum[2][MAX_MATRIX_SIZE*2];
	//get MWB executin time

	int start = 1;
	int end=32;
	for (n = start; n <= end; n += 1)
	{

		double **A;//���͔z��
		double **wr;
		double **wi;
		A = (double**)malloc(sizeof(double*)*batchCount);
		wr = (double**)malloc(sizeof(double*)*batchCount);
		wi = (double**)malloc(sizeof(double*)*batchCount);
		
		
		for (i = 0; i < batchCount; i++)
		{
			A[i] = (double*)malloc(sizeof(double)*n*n);
			wr[i] = (double*)malloc(sizeof(double)*n);
			wi[i] = (double*)malloc(sizeof(double)*n);
		}
		set_mat(A, n, batchCount);
		for (i = 0; i < batchCount; i++)
		{
			int j;
			for (j = 0; j < n; j++)
			{
				wr[i][j] = 0.0;
				wi[i][j] = 0.0;
			}

		}


		int threads=tokura_dgeev_batched_MWBtune(n, A, wr, wi, batchCount,time[0]);
		threadnum[0][n] = threads & 0xffff;
		threadnum[0][n+ MAX_MATRIX_SIZE] = (threads >>16)& 0xffff;

		for (i = 0; i < batchCount; i++)
		{
			free(A[i]);
			free(wr[i]);
			free(wi[i]);
		}
		free(A);
		free(wr);
		free(wi);

		printf("MWB:%d end\n", n);
	}
	//get SWB executin time
	for (n = start; n <= end; n += 1)
	{

		double **A;//���͔z��
		double **wr;
		double **wi;
		A = (double**)malloc(sizeof(double*)*batchCount);
		wr = (double**)malloc(sizeof(double*)*batchCount);
		wi = (double**)malloc(sizeof(double*)*batchCount);


		for (i = 0; i < batchCount; i++)
		{
			A[i] = (double*)malloc(sizeof(double)*n*n);
			wr[i] = (double*)malloc(sizeof(double)*n);
			wi[i] = (double*)malloc(sizeof(double)*n);
		}
		set_mat(A, n, batchCount);
		for (i = 0; i < batchCount; i++)
		{
			int j;
			for (j = 0; j < n; j++)
			{
				wr[i][j] = 0.0;
				wi[i][j] = 0.0;
			}

		}

		threadnum[1][n] = tokura_dgeev_batched_SWBtune(n, A, wr, wi, batchCount, time[1]);

		for (i = 0; i < batchCount; i++)
		{
			free(A[i]);
			free(wr[i]);
			free(wi[i]);
		}
		free(A);
		free(wr);
		free(wi);

		printf("SWB:%d end\n", n);

	}

	for (n = start; n <= end; n += 1)
	{
		printf("%d:%lf %lf[ms]\n", n, time[0][n], time[1][n]);

	}

	

	parameter_write(time, threadnum);
	return 0;
}
