#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"tokura_blas_define.h"
#include"tokura_blas_const.h"

//This kernel rearrenge matrix arrangement from Matrix wise to Elemental wise.
__global__ void tokura_matrixrearrangement_MWtoEW(const int n, const double* __restrict__ a, double *a2, const int batchCount_per_stream)
{
	const int tmp_mat_pad = blockIdx.x * 32;
	const int shared_pad = 32 + 1;
	const int n_n = n*n;

	if (!(tmp_mat_pad < batchCount_per_stream))
	{
		return;
	}


	const int remain_mat = (tmp_mat_pad + 32) < batchCount_per_stream ? 32 : (batchCount_per_stream - tmp_mat_pad);


	int i, j, k;
	int remain = 0;
	int tmp_remain;

	__shared__ double tmp_mat_shared[32 * shared_pad];


	while ((remain) < n_n)
	{
		tmp_remain = ((remain + 32) < n_n) ? 32 : (n_n - remain);

		if ((remain + (threadIdx.x&(32 - 1))) < n_n)
		{
			for (k = 0 + (threadIdx.x >> 5); k < remain_mat; k += (blockDim.x >> 5))
			{
				tmp_mat_shared[(threadIdx.x&(32 - 1)) + k*shared_pad] = a[remain + (threadIdx.x&(32 - 1)) + (tmp_mat_pad + k)*n_n];
			}
		}
		__syncthreads();

		if ((threadIdx.x&(32 - 1)) < remain_mat)
		{
			for (k = 0 + (threadIdx.x >> 5); k < tmp_remain; k += (blockDim.x >> 5)/*k++*/)
			{
				j = (remain + k) / n;
				i = (remain + k) % n;
			

				a2[(/*(remain + k)*/j + i*n)*batchCount_per_stream + (tmp_mat_pad + (threadIdx.x&(32 - 1)))] = tmp_mat_shared[k + ((threadIdx.x&(32 - 1)))*shared_pad];
			}
		}

		remain += 32;
		__syncthreads();

	}
}


//This kernel rearrenge eigenvalues arrangement from Elemental  wise to Matrix wise.
__global__ void tokura_eigenvaluesrearrangement_EWtoMW(const int n, const  int mat_num, const double* __restrict__ comp_real, const double* __restrict__ comp_imag, double *comp_real_out, double *comp_imag_out)
{
	const int tmp_mat_pad = blockIdx.x * 32;
	const int shared_pad = 32 + 1;
	const int n_n = n*n;

	if (!(tmp_mat_pad < mat_num))
	{
		return;
	}


	const int remain_mat = (tmp_mat_pad + 32) < mat_num ? 32 : (mat_num - tmp_mat_pad);


	int i, j, k;
	int remain = 0;
	int tmp_remain;

	extern __shared__ double shared_d_eig_MWB[];
	double *real_s = &shared_d_eig_MWB[0];
	double *imag_s = &shared_d_eig_MWB[n*shared_pad];


	if (threadIdx.x < remain_mat)
	{
		for (i = threadIdx.y; i < n; i += blockDim.y)
		{
			real_s[threadIdx.x + i*shared_pad] = comp_real[i*mat_num + tmp_mat_pad + threadIdx.x];
			imag_s[threadIdx.x + i*shared_pad] = comp_imag[i*mat_num + tmp_mat_pad + threadIdx.x];
		}

	}
	__syncthreads();

	if (threadIdx.x < n)
	{
		for (i = threadIdx.y; i < remain_mat; i += blockDim.y)
		{
			comp_real_out[threadIdx.x + (tmp_mat_pad + i)*n] = real_s[i + threadIdx.x*shared_pad];
			comp_imag_out[threadIdx.x + (tmp_mat_pad + i)*n] = imag_s[i + threadIdx.x*shared_pad];
		}
	}


}




//This kernel rearrenge matrix arrangement from Matrix wise to Row wise.
__global__ void tokura_matrixrearrangement_RWtoEW
(
	const int n, 
	const  int mat_num,
	const double* __restrict__ a, 
	double *a2
)
{
	const int tmp_mat_pad = blockIdx.x * 32;
	const int shared_pad = 32 + 1;
	const int n_n = n*n;



	const int remain_mat = (tmp_mat_pad + 32) <= mat_num ? 32 : (mat_num - tmp_mat_pad);


	int i, j, k;
	int remain = 0;
	int tmp_remain;


	__shared__ double tmp_mat_shared[32 * shared_pad];


	if (threadIdx.x < n)
	{
		for (i = threadIdx.y; i <remain_mat; i += blockDim.y)
		{
			tmp_mat_shared[i + threadIdx.x * shared_pad] = a[(tmp_mat_pad + i)*n + threadIdx.x + (blockIdx.y)*mat_num*n];
		}
	}

	__syncthreads();

	if (threadIdx.x< remain_mat)
	{
		for (i = threadIdx.y; i < n; i += blockDim.y)
		{

			a2[(blockIdx.y + i*n)*mat_num + tmp_mat_pad + threadIdx.x] = tmp_mat_shared[threadIdx.x + i * shared_pad];

		}
	}
	__syncthreads();
}


//This kernel rearrenge matrix arrangement from Matrix wise to Row wise.
__global__ void tokura_matrixrearrangement_MWtoRW
(
	int n,
	double *input,
	double *output,
	int mat_num
)
{

	const int shared_pad = 32 + 1;
	__shared__ double tmp_mat_shared[32 * shared_pad];

	int i;
	if (threadIdx.x < n)
	{
		for (i = threadIdx.y; i < n; i += blockDim.y)
		{
			tmp_mat_shared[threadIdx.x*shared_pad + i] = input[((i)*n + (threadIdx.x)) + blockIdx.x*n*n];
		}
	}

	__syncthreads();

	if (threadIdx.x < n)
	{
		for (i = threadIdx.y; i < n; i += blockDim.y)
		{
			output[threadIdx.x + i*mat_num*n + blockIdx.x*n] = tmp_mat_shared[threadIdx.x + i*shared_pad];
		}
	}
}

