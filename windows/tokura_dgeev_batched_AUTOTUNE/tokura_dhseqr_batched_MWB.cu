#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"tokura_blas_define.h"
#include"tokura_blas_const.h"






///*���Ίp�v�f�`�F�b�N,�o���W����,�o���W�`�F�[�V���O����*/
//DHSEQR computes eigenvalues of Hessenberg matrix
//Any eigenvectors are NOT computed.
__global__ void tokura_dhseqr_batched_MWB(const int n, const int mat_num, double*  a, const int DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR, double *comp_real, double *comp_imag)
{

	const int matrix_id = threadIdx.x + threadIdx.z*blockDim.x;
	const int warpid = threadIdx.y;
	const int mat_pad = (blockIdx.x*blockDim.x*blockDim.z) + matrix_id;
	const int MATRIX_NUM_PER_BLOCK = blockDim.x*blockDim.z;
	if (!((mat_pad < mat_num)))
	{
		return;
	}

	int nn, m, j, its, i, mmin;
	double /*z, y, x,*/ r, q, p;
	double tmp = 0.0;
	double tau = 0.0;
	int k;


	int ZERO_FLAG = 0;
	extern __shared__ int dynamic_shared[];

	int *zero_index = &dynamic_shared[0];/*���Ίp�v�f��0�ł���C���f�b�N�X(��)���i�[*/
	int *eig_num_shared = &zero_index[(n + 1)*MATRIX_NUM_PER_BLOCK];

	if (warpid == 0)
	{
		//zero_index[0][matrix_id] = 0;
		zero_index[matrix_id + (0 * MATRIX_NUM_PER_BLOCK)] = 0;
		//zero_index[n][matrix_id] = n;
		zero_index[matrix_id + (n *MATRIX_NUM_PER_BLOCK)] = n;
	}


	nn = n;
	its = 0;
	i = 1 + warpid;
	while (i < n /*+ 1*/)
	{
		zero_index[matrix_id + (i*MATRIX_NUM_PER_BLOCK)] = 0;
		i += DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;
	}

	__syncthreads();
	//	__syncwarp();
	while (1/*its != 100000*/)
	{


		/*�s����S�~�̒l�����J�n*/
		i = 2 + warpid;
		while (i < n)
		{
			a[((i - 2) + i*n)*mat_num + mat_pad] = 0.0;
			i += DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;
		}
		i = 3 + warpid;
		while (i < n)
		{
			a[((i - 3) + i*n)*mat_num + mat_pad] = 0.0;
			i += DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;
		}
		__syncthreads();
		//	__syncwarp();

		m = (n - 1) - warpid;
		while (0 < m)
		{
			if (zero_index[matrix_id + (m*MATRIX_NUM_PER_BLOCK)] == 0)
			{
				tmp = fabs(a[(m + m*n)*mat_num + mat_pad]) + fabs(a[((m - 1) + (m - 1)*n)*mat_num + mat_pad]);
				if (tmp == 0)
				{
					tmp = 1.0;
				}

				if ((fabs(a[((m - 1) + m*n)*mat_num + mat_pad]) + tmp) == tmp)
				{
					zero_index[matrix_id + (m*MATRIX_NUM_PER_BLOCK)] = m;
				}
			}
			m -= DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;

		}
		//__syncwarp();

		__syncthreads();
		if (warpid == 0)
		{
			/*�ϐ����蓖�ĕύX
			eig_num->i*/
			//eig_num = 0;
			i = 0;
			/*�ϐ����蓖�ĕύX
			EIG_END->j*/
			//EIG_END = n;
			j = n;
			for (m = n - 1; 0 < m; m--)
			{
				if (zero_index[matrix_id + (m *MATRIX_NUM_PER_BLOCK)] != 0)
				{
					a[((m - 1) + m*n)*mat_num + mat_pad] = 0.0;
					nn = j - m;
					j = m;
					if (nn < 3)
					{
						i += nn;
					}
				}
			}

			nn = j - 0;
			if (nn < 3)
			{
				i += nn;
			}
			eig_num_shared[matrix_id] = i;
		}
		__syncthreads();
		//__syncwarp();



		/*if (eig_num_shared[matrix_id] == n)
		{

		break;
		}*/
		unsigned int tmp_flag_ballot = eig_num_shared[matrix_id] == n;
		__syncthreads();
		//__syncwarp();
		unsigned int ballot_mask;
		ballot_mask = __ballot_sync(__activemask(), tmp_flag_ballot);

		if (ballot_mask == __activemask())
		{
			break;
		}
		its++;

		__syncthreads();
		//bulge generating 
		{
			m = 0;
			nn = n;

			k = 0;
			for (m = 0; m + 2 < n; m++)
			{
				nn = n;

				/*ZERO_FLAG==1�̂Ƃ��̂݃o���W���ł���*/
				ZERO_FLAG = ((zero_index[matrix_id + (m *MATRIX_NUM_PER_BLOCK)] != 0) || (m == 0)) && ((zero_index[matrix_id + ((m + 1) *MATRIX_NUM_PER_BLOCK)] == 0) && (zero_index[matrix_id + ((m + 2) *MATRIX_NUM_PER_BLOCK)] == 0));
				//	__syncwarp();

				if (ZERO_FLAG == 1)
				{
					for (i = m + 3; i < n + 1; i++)
					{
						if (zero_index[matrix_id + (i*MATRIX_NUM_PER_BLOCK)] != 0)
						{
							nn = zero_index[matrix_id + (i *MATRIX_NUM_PER_BLOCK)];
							break;
						}
					}
				}
				//	__syncwarp();
				if (ZERO_FLAG == 1)
				{

					//}



					if (warpid < (nn - m/*k*/))
					{
						/*�o���W�����J�n*/
						//	if (ZERO_FLAG == 1)
						//	{
						/*�V�t�g�̑I��*/
						if (its % 10 == 0)
						{
							p = fabs(a[(((nn - 1 - 1) + (nn - 1)*n))*mat_num + mat_pad]);
							p = p + fabs(a[(((nn - 2 - 1) + (nn - 1 - 1)*n))*mat_num + mat_pad]);
							r = 1.5*(p);
							tmp = p*p;
						}
						else
						{
							p = a[((nn - 1) + (nn - 1)*n)*mat_num + mat_pad];
							q = a[((nn - 1 - 1) + (nn - 1 - 1)*n)*mat_num + mat_pad];
							r = p + q;
							tmp = p*q - a[((nn - 1 - 1) + (nn - 1)*n)*mat_num + mat_pad] * a[((nn - 1) + (nn - 1 - 1)*n)*mat_num + mat_pad]/*w*/;
						}

					}
				}


				//	__syncwarp();
				if (ZERO_FLAG == 1)
				{

					ZERO_FLAG = ZERO_FLAG && (a[((m)+(m + 1)*n)*mat_num + mat_pad] != 0.0);
				}

				//	__syncwarp();
				__syncthreads();

				/*�n�E�X�z���_�[�ϊ��\�z�J�n*/
				if (ZERO_FLAG == 1)
				{
					if (warpid < (nn - m/*k*/))
					{
						p = (a[(((m)+(m)*n))*mat_num + mat_pad] * (a[(((m)+(m)*n))*mat_num + mat_pad] - r) + tmp) / a[((m)+(m + 1)*n)*mat_num + mat_pad] + a[((m + 1) + (m)*n)*mat_num + mat_pad];
						q = a[(((m)+(m)*n))*mat_num + mat_pad] + a[((m + 1) + (m + 1)*n)*mat_num + mat_pad] - r;
						r = a[((m + 1) + (m + 2)*n)*mat_num + mat_pad];


						/*���x�m�ۂ̂��߂̏����J�n*/
						tmp = fabs(p) + fabs(q) + fabs(r);
						p /= tmp;
						q /= tmp;
						r /= tmp;
						/*���x�m�ۂ̂��߂̏����I��*/

						tau = q*q + r*r;
						tmp = p*p + tau;
						tmp = sqrt(fabs(tmp));

						p += (p > 0.0 ? tmp : -tmp);
						tmp = p*p + tau;
						tau = 2.0 / tmp;
						/*x = tmp*p;
						y = tmp*q;
						z = tmp*r;*/

						/*E-[x;y;z][p q r]*/
						/*�n�E�X�z���_�[�ϊ��\�z�I��*/
			}

		}
				__syncthreads();
				//	__syncwarp();

#if USE_DOUBLE==0		
				if (isnan(tau) || isnan(p) || isnan(q) || isnan(r))
				{
					a[((m)+(m + 1)*n)*mat_num + mat_pad] = 0.0;
					continue;
				}
				if (isinf(tau) || isinf(p) || isinf(q) || isinf(r))
				{
					a[((m)+(m + 1)*n)*mat_num + mat_pad] = 0.0;
					continue;
				}
#endif
				if (ZERO_FLAG == 1)
				{
					/*�o���W�����J�n*/
					j = m + warpid;
					if (warpid < (nn - m/*k*/))
					{
						while (j < nn)
						{
							tmp = p*a[(j + (m + 0)*n)*mat_num + mat_pad] + q*a[(j + (m + 1)*n)*mat_num + mat_pad] + r*a[(j + (m + 2)*n)*mat_num + mat_pad];
							tmp *= tau;
							a[(j + (m + 2)*n)*mat_num + mat_pad] -= r*tmp;
							a[(j + (m + 1)*n)*mat_num + mat_pad] -= q*tmp;
							a[(j + (m + 0)*n)*mat_num + mat_pad] -= p*tmp;
							j += DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;
						}
					}
					mmin = ((nn < (m + 3 + 1)) ? nn : m + 3 + 1);
				}

				{
					__syncthreads();
					//	__syncwarp();

				}

				if (ZERO_FLAG == 1)
				{
					//for (i = m; i < mmin; i++)
					i = m + warpid;
					if (warpid < (nn - m/*k*/))
					{
						while (i < mmin)
						{
							tmp = p*a[((m + 0) + i*n)*mat_num + mat_pad] + q*a[((m + 1) + i*n)*mat_num + mat_pad] + r*a[((m + 2) + i*n)*mat_num + mat_pad];
							tmp *= tau;
							a[((m + 2) + i*n)*mat_num + mat_pad] -= r*tmp;
							a[((m + 1) + i*n)*mat_num + mat_pad] -= q*tmp;
							a[((m + 0) + i*n)*mat_num + mat_pad] -= p*tmp;
							i += DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;
						}
					}
					/*�o���W�����I��*/
					k = nn;
					//m = nn - 1;
				}
				{
					__syncthreads();
					//	__syncwarp();

				}


				//m = nn;/*���̃o���W�����*/

	}

}

		__syncthreads();
		//return;

		//bulge chasing
		nn = n;
		k = 0;
		for (m = 0; m + 1 + 2 < n + 1/*m < nn-1-1*/; m++)
		{


			if (zero_index[matrix_id + ((m + 1) *MATRIX_NUM_PER_BLOCK)] != 0)
			{
				k = m + 1;

			}
			else if (zero_index[matrix_id + ((m + 1 + 1)*MATRIX_NUM_PER_BLOCK)] != 0)
			{
				k = m + 1 + 1;
			}
			//__syncwarp();

			ZERO_FLAG = (zero_index[matrix_id + ((m + 1) *MATRIX_NUM_PER_BLOCK)] == 0) && (zero_index[matrix_id + ((m + 1 + 1) *MATRIX_NUM_PER_BLOCK)] == 0);

			if (ZERO_FLAG == 1)
			{
				/*�o���W�𐶐��ł��邩���f*/
				for (i = m + 1 + 1 + 1; i < n + 1; i++)
				{
					if (zero_index[matrix_id + (i*MATRIX_NUM_PER_BLOCK)] != 0)
					{
						nn = zero_index[matrix_id + (i *MATRIX_NUM_PER_BLOCK)];

						break;
					}
				}
				//}
			}
			//	__syncwarp();
			if (ZERO_FLAG == 1)
			{


				if (warpid < (nn - k))
				{
					//	if (ZERO_FLAG == 1)
					//	{
					p = a[(m + (m + 1 + 0)*n)*mat_num + mat_pad];
					q = a[(m + (m + 1 + 1)*n)*mat_num + mat_pad];
					if ((m + 1 + 2) < nn)
					{
						r = a[(m + (m + 1 + 2)*n)*mat_num + mat_pad];
					}
					else
					{
						r = 0.0;
					}

					tau = q*q + r*r;

					tmp = p*p + tau;
					tmp = sqrt(fabs(tmp));


					p += ((p > 0.0) ? tmp : -tmp);


					tmp = p*p + tau;


					if ((ZERO_FLAG == 1) && (tmp != 0.0))
					{
						tau = 2.0 / tmp;

					}
					ZERO_FLAG = ZERO_FLAG && (tmp != 0.0);
				}
					}
			__syncthreads();
			//__syncwarp();

			if ((ZERO_FLAG == 1))
			{

				j = m + warpid;
				if (warpid < (nn - k))
				{
					while (j < nn)
					{

						tmp = p*a[(j + (m + 1 + 0)*n)*mat_num + mat_pad] + q*a[(j + (m + 1 + 1)*n)*mat_num + mat_pad];
						tmp *= tau;
						if ((m + 1 + 2) < nn)
						{
							tmp += tau*r*a[(j + (m + 1 + 2)*n)*mat_num + mat_pad];
							a[(j + (m + 1 + 2)*n)*mat_num + mat_pad] -= r*tmp;
						}
						a[(j + (m + 1 + 1)*n)*mat_num + mat_pad] -= q*tmp;
						a[(j + (m + 1 + 0)*n)*mat_num + mat_pad] -= p*tmp;

						j += DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;
					}
				}
			}
			__syncwarp();

			mmin = (nn < (m + 3 + 1 + 1) ? nn : (m + 3 + 1 + 1));

			__syncthreads();
		

			if ((ZERO_FLAG == 1))
			{

				i = k + warpid;
				if (warpid < (nn - k))
				{
					while (i < mmin)
					{

						tmp = p*a[((m + 1 + 0) + i*n)*mat_num + mat_pad] + q*a[((m + 1 + 1) + i*n)*mat_num + mat_pad];
						tmp *= tau;
						if ((m + 1 + 2) < nn)
						{
							tmp += tau*r*a[((m + 1 + 2) + i*n)*mat_num + mat_pad];
							a[((m + 1 + 2) + i*n)*mat_num + mat_pad] -= r*tmp;
						}
						a[((m + 1 + 1) + i*n)*mat_num + mat_pad] -= q*tmp;
						a[((m + 1 + 0) + i*n)*mat_num + mat_pad] -= p*tmp;

						i += DOUBLE_QR_KERNEL_THREAD_JOIN_NUM_VAR;
					}
				}
			}
			__syncthreads();

				}
		__syncthreads();

			}

	//Eigenvalues of 1*1 or 2*2 computation
	//Stride access may be pefrmed at eigenvalue storing to global memory
	//However, the overhead is much lower than above computation.
	//So, the overhead can be ignored.
	if (warpid == 0)
	{
		{
			double x, y, z;
			nn = n - 1;
			while (nn >= 0)
			{
				m = (zero_index[matrix_id + (nn *MATRIX_NUM_PER_BLOCK)] != 0) || (nn == 0) ? nn : nn - 1;
				if (m == nn)
				{
					comp_real[nn*mat_num + mat_pad] = a[(nn + nn*n)*mat_num + mat_pad];
					comp_imag[nn*mat_num + mat_pad] = 0.0;		
					nn--;
				}
				else
				{
					tau = a[((nn - 1) + (nn - 1)*n)*mat_num + mat_pad];
					p = 0.5*(a[(nn + nn*n)*mat_num + mat_pad] + tau/*y*/);
					q = 4.0*(a[((nn - 1) + nn*n)*mat_num + mat_pad] * a[(nn + (nn - 1)*n)*mat_num + mat_pad]) + (a[(nn + nn*n)*mat_num + mat_pad]/*x*/ - tau/*y*/)*(a[(nn + nn*n)*mat_num + mat_pad]/*x*/ - tau/*y*/);
					tmp = 0.5*sqrt(fabs(q));

					comp_real[nn*mat_num + mat_pad] = p + ((q >= 0.0) ? tmp/*z*/ : 0.0);
					comp_imag[nn*mat_num + mat_pad] = q < 0.0 ? tmp/*z*/ : 0.0;
					comp_real[(nn - 1)*mat_num + mat_pad] = p - ((q >= 0.0) ? tmp/*z*/ : 0.0);
					comp_imag[(nn - 1)*mat_num + mat_pad] = q < 0.0 ? -tmp/*-z*/ : 0.0;
					nn -= 2;
				}
			}
		}
	}


}
