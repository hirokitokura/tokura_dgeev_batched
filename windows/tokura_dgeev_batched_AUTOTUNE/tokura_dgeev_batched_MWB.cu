
#include "hip/hip_runtime.h"
#include ""
//
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"tokura_blas_define.h"
#include"tokura_blas_const.h"

#include"tokura_blas.h"
#include"tokura_blas_functions.h"

#include<float.h>



int tokura_dgeev_batched_MWB_tune(tokuraInternalhandle_t* tokurahandle, int n, double** A, double** wr, double** wi, int batchCount, float* time)
{
	int i, j;
	int matrix_index;
	int batchCount_per_stream;
	int WARPSIZE = tokurahandle->WARP_SIZE;

	int hessen_join_num;
	int doubleqr_join_num;
	int stream_id;


	hessen_join_num = get_hessenbergreduction_MWBthreads_num(n);
	doubleqr_join_num = get_doubleshiftQR_MWBthreads_num(n);
	hipEvent_t start, stop;
	float elapsed_time_ms = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	matrix_index = 0;
	float hrd_local_time = FLT_MAX;
	float qr_local_time=FLT_MAX;
	i = 0;
	int optimal_hrd_thread=0;
	int optimal_qr_thread=0;
	int thread_count;
	for(thread_count=1;thread_count<=n;thread_count++)
	{
	
		hessen_join_num = thread_count;
		doubleqr_join_num = thread_count;
			stream_id = 0;

			batchCount_per_stream= batchCount;

			
				for (int transferedmatrixid = 0; transferedmatrixid < batchCount_per_stream; transferedmatrixid++)
				{
					hipMemcpyAsync
					(
						&tokurahandle->A_tmp_device[stream_id][(matrix_index + transferedmatrixid)*n*n],
						A[matrix_index + transferedmatrixid],
						sizeof(double)*n*n,
						hipMemcpyHostToDevice,
						tokurahandle->stream[stream_id]

					);

				}
				hipDeviceSynchronize();

				hipEventRecord(start, 0);

				tokura_matrixrearrangement_MWtoEW
					<< <
					(batchCount_per_stream + WARPSIZE - 1) / WARPSIZE,
					TRANSPOSE_CUDA_THREADS_MULTI,
					0
					>> >
					(
						n,
						tokurahandle->A_tmp_device[stream_id],
						tokurahandle->A_device[stream_id],
						batchCount_per_stream
						);

				dim3 thread_hessen(WARPSIZE, hessen_join_num);
				dim3 block_hessen((batchCount_per_stream + (WARPSIZE)-1) / (WARPSIZE));
				tokura_dgehrd_batched_MWB << <block_hessen, thread_hessen, sizeof(double)*(n - 1) * WARPSIZE + sizeof(double)*(hessen_join_num)* WARPSIZE >> >
					(
						n,
						batchCount_per_stream,
						tokurahandle->A_device[stream_id],
						hessen_join_num
						);


				hipDeviceSynchronize();
				
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);

				hipEventElapsedTime(&elapsed_time_ms, start, stop);
				if (hrd_local_time > elapsed_time_ms)
				{
					hrd_local_time = elapsed_time_ms;
					optimal_hrd_thread = thread_count;
				}

				hipEventRecord(start, 0);
				dim3 thread_qr(WARPSIZE, doubleqr_join_num);
				dim3 block_qr((batchCount_per_stream + (WARPSIZE)-1) / (WARPSIZE));
				tokura_dhseqr_batched_MWB << <block_qr, thread_qr, sizeof(int)*(n + 1) * (WARPSIZE)+sizeof(int)* (WARPSIZE)>> >
					(
						n,
						batchCount_per_stream,
						tokurahandle->A_device[stream_id],
						doubleqr_join_num,
						tokurahandle->wr_device[stream_id],
						tokurahandle->wi_device[stream_id]
						);
				hipDeviceSynchronize();
				
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);

				hipEventElapsedTime(&elapsed_time_ms, start, stop);
				if (qr_local_time > elapsed_time_ms)
				{
					qr_local_time = elapsed_time_ms;
					optimal_qr_thread = thread_count;
				}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	time[n] = hrd_local_time;
	//time[n+32] = qr_local_time;
	return (optimal_qr_thread << 16) + optimal_hrd_thread;
}

