
#include "hip/hip_runtime.h"
#include ""
//
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"tokura_blas_define.h"
#include"tokura_blas_const.h"
//#include"tokura_tuned_thread_parameters.h"
int get_hessenbergreduction_MWBthreads_num(int n)
{
	int parameter;
	switch (n)
	{
/*	case 1:
		parameter = TOKURA_MWB_HRD_1;
		break;
	case 2:
		parameter = TOKURA_MWB_HRD_2;
		break;
	case 3:
		parameter = TOKURA_MWB_HRD_3;
		break;
	case 4:
		parameter = TOKURA_MWB_HRD_4;
		break;
	case 5:
		parameter = TOKURA_MWB_HRD_5;
		break;
	case 6:
		parameter = TOKURA_MWB_HRD_6;
		break;
	case 7:
		parameter = TOKURA_MWB_HRD_7;
		break;
	case 8:
		parameter = TOKURA_MWB_HRD_8;
		break;
	case 9:
		parameter = TOKURA_MWB_HRD_9;
		break;
	case 10:
		parameter = TOKURA_MWB_HRD_10;
		break;
	case 11:
		parameter = TOKURA_MWB_HRD_11;
		break;
	case 12:
		parameter = TOKURA_MWB_HRD_12;
		break;
	case 13:
		parameter = TOKURA_MWB_HRD_13;
		break;
	case 14:
		parameter = TOKURA_MWB_HRD_14;
		break;
	case 15:
		parameter = TOKURA_MWB_HRD_15;
		break;
	case 16:
		parameter = TOKURA_MWB_HRD_16;
		break;
	case 17:
		parameter = TOKURA_MWB_HRD_17;
		break;
	case 18:
		parameter = TOKURA_MWB_HRD_18;
		break;
	case 19:
		parameter = TOKURA_MWB_HRD_19;
		break;
	case 20:
		parameter = TOKURA_MWB_HRD_20;
		break;
	case 21:
		parameter = TOKURA_MWB_HRD_21;
		break;
	case 22:
		parameter = TOKURA_MWB_HRD_22;
		break;
	case 23:
		parameter = TOKURA_MWB_HRD_23;
		break;
	case 24:
		parameter = TOKURA_MWB_HRD_24;
		break;
	case 25:
		parameter = TOKURA_MWB_HRD_25;
		break;
	case 26:
		parameter = TOKURA_MWB_HRD_26;
		break;
	case 27:
		parameter = TOKURA_MWB_HRD_27;
		break;
	case 28:
		parameter = TOKURA_MWB_HRD_28;
		break;
	case 29:
		parameter = TOKURA_MWB_HRD_29;
		break;
	case 30:
		parameter = TOKURA_MWB_HRD_30;
		break;
	case 31:
		parameter = TOKURA_MWB_HRD_31;
		break;
	case 32:
		parameter = TOKURA_MWB_HRD_32;
		break;*/
	default:
		parameter = n;
		break;
	}
	return parameter;
}

int get_hessenbergreduction_SWBthreads_num(int n)
{
	int parameter;
	switch (n)
	{
/*	case 1:
		parameter = TOKURA_SWB_HRD_1;
		break;
	case 2:
		parameter = TOKURA_SWB_HRD_2;
		break;
	case 3:
		parameter = TOKURA_SWB_HRD_3;
		break;
	case 4:
		parameter = TOKURA_SWB_HRD_4;
		break;
	case 5:
		parameter = TOKURA_SWB_HRD_5;
		break;
	case 6:
		parameter = TOKURA_SWB_HRD_6;
		break;
	case 7:
		parameter = TOKURA_SWB_HRD_7;
		break;
	case 8:
		parameter = TOKURA_SWB_HRD_8;
		break;
	case 9:
		parameter = TOKURA_SWB_HRD_9;
		break;
	case 10:
		parameter = TOKURA_SWB_HRD_10;
		break;
	case 11:
		parameter = TOKURA_SWB_HRD_11;
		break;
	case 12:
		parameter = TOKURA_SWB_HRD_12;
		break;
	case 13:
		parameter = TOKURA_SWB_HRD_13;
		break;
	case 14:
		parameter = TOKURA_SWB_HRD_14;
		break;
	case 15:
		parameter = TOKURA_SWB_HRD_15;
		break;
	case 16:
		parameter = TOKURA_SWB_HRD_16;
		break;
	case 17:
		parameter = TOKURA_SWB_HRD_17;
		break;
	case 18:
		parameter = TOKURA_SWB_HRD_18;
		break;
	case 19:
		parameter = TOKURA_SWB_HRD_19;
		break;
	case 20:
		parameter = TOKURA_SWB_HRD_20;
		break;
	case 21:
		parameter = TOKURA_SWB_HRD_21;
		break;
	case 22:
		parameter = TOKURA_SWB_HRD_22;
		break;
	case 23:
		parameter = TOKURA_SWB_HRD_23;
		break;
	case 24:
		parameter = TOKURA_SWB_HRD_24;
		break;
	case 25:
		parameter = TOKURA_SWB_HRD_25;
		break;
	case 26:
		parameter = TOKURA_SWB_HRD_26;
		break;
	case 27:
		parameter = TOKURA_SWB_HRD_27;
		break;
	case 28:
		parameter = TOKURA_SWB_HRD_28;
		break;
	case 29:
		parameter = TOKURA_SWB_HRD_29;
		break;
	case 30:
		parameter = TOKURA_SWB_HRD_30;
		break;
	case 31:
		parameter = TOKURA_SWB_HRD_31;
		break;
	case 32:
		parameter = TOKURA_SWB_HRD_32;
		break;*/
	default:
		parameter = n;
		break;
	}
	return parameter;


}
int get_doubleshiftQR_MWBthreads_num(int n)
{
	int parameter;
	switch (n)
	{
	/*case 1:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_1;
		break;
	case 2:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_2;
		break;
	case 3:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_3;
		break;
	case 4:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_4;
		break;
	case 5:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_5;
		break;
	case 6:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_6;
		break;
	case 7:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_7;
		break;
	case 8:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_8;
		break;
	case 9:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_9;
		break;
	case 10:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_10;
		break;
	case 11:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_11;
		break;
	case 12:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_12;
		break;
	case 13:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_13;
		break;
	case 14:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_14;
		break;
	case 15:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_15;
		break;
	case 16:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_16;
		break;
	case 17:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_17;
		break;
	case 18:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_18;
		break;
	case 19:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_19;
		break;
	case 20:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_20;
		break;
	case 21:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_21;
		break;
	case 22:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_22;
		break;
	case 23:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_23;
		break;
	case 24:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_24;
		break;
	case 25:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_25;
		break;
	case 26:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_26;
		break;
	case 27:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_27;
		break;
	case 28:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_28;
		break;
	case 29:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_29;
		break;
	case 30:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_30;
		break;
	case 31:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_31;
		break;
	case 32:
		parameter = TOKURA_MWB_DOUBLESHIFTQR_32;
		break;*/
	default:
		parameter = n;
		break;
	}
	return parameter;
}
